#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "template.hu"

#define TILE_SZ_A 128
#define TILE_SZ_B 16
#define TILE_SZ_RATIO (TILE_SZ_A/TILE_SZ_B)

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

  /********************************************************************
  *
  * Compute C = A x B
  *   where A is a (m x k) matrix
  *   where B is a (k x n) matrix
  *   where C is a (m x n) matrix
  *
  * Use register and shared memory tiling and thread coarsening
  *
  * NOTE: A and C are column major, B is row major
  *
  ********************************************************************/

  // Macros for accessing flattened matrices
  #define A(row,col) A[(row) + (col)*m]
  #define B(row,col) B[(row)*n + (col)]
  #define C(row,col) C[(row) + (col)*m]

  // INSERT KERNEL CODE HERE
	__shared__ float MatBCache[TILE_SZ_RATIO][TILE_SZ_B];
  // thread local array for partial sums
  float pvalues[TILE_SZ_B];
  for (int i=0; i<TILE_SZ_B; i++){
    pvalues[i] = 0;
  }
	int ty = threadIdx.y;
	int Row = blockIdx.y * blockDim.y + ty;
	int Col_Start = blockIdx.x * TILE_SZ_B;
	int Col_End = Col_Start + TILE_SZ_B - 1;
	Col_End = (Col_End > n-1)? n-1 : Col_End;
  int numCols = Col_End - Col_Start + 1;
	int numIteration = (k - 1)/TILE_SZ_RATIO + 1;
	for (int q = 0; q < numIteration; q++){
		int cache_i = ty / TILE_SZ_B;
		int cache_j = ty % TILE_SZ_B;
		int MatB_Row_Idx = q*TILE_SZ_RATIO + cache_i;
		int MatB_Col_Idx = Col_Start + cache_j;
		if (MatB_Col_Idx < n && MatB_Row_Idx < k){
			MatBCache[cache_i][cache_j] = B(MatB_Row_Idx, MatB_Col_Idx);
		}else{
      MatBCache[cache_i][cache_j] = 0;
    }
		float MatATile_0 = (Row < m)? A(Row, q*TILE_SZ_RATIO): 0;
		__syncthreads();
		if (Row < m){
			for (int Col = Col_Start; Col <= Col_End; Col++){
				int Col_relative = Col - Col_Start;
				// float pvalue = MatATile_0 * MatBCache[0][Col_relative];
				// for (int s = 1; s < TILE_SZ_RATIO; s++){
				// 	pvalue += (MatBCache[s][Col_relative] * A(Row, q*TILE_SZ_RATIO + s));
				// }
        // pvalues[Col_relative] += pvalue;
        pvalues[Col_relative] += MatATile_0 * MatBCache[0][Col_relative];
				for (int s = 1; s < TILE_SZ_RATIO; s++){
					pvalues[Col_relative] += (MatBCache[s][Col_relative] * A(Row, q*TILE_SZ_RATIO + s));
				}
			}
		}
    __syncthreads();
	}
  if (Row < m){
      for (int i=0; i<numCols; i++){
        C(Row, Col_Start+i) = pvalues[i];
      }
  }
  // SSL Hint (9/6/21): try using just one register for the tile of A 
  // rather than several--in other words, load one value (per thread) 
  // from A and compute using that value rather than loading all values 
  // before doing the computation.  This approach seems to be slightly 
  // faster than the alternative.
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
  if ((transa != 'N') && (transa != 'n')) {
		printf("unsupported value of 'transa'\n");
    return;
  }

  if ((transb != 'T') && (transb != 't')) {
		printf("unsupported value of 'transb'\n");
		return;
  }

  if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
		printf("unsupported value of alpha\n");
		return;
  }

  if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
		printf("unsupported value of beta\n");
		return;
  }

	// Initialize thread block and kernel grid dimensions ---------------------

	// Your code need only consider the m, n, k, A, B, and C parameters of
	// the function, which provide the matrix sizes (m, n, k) and data
	// (A, B, C).

	//INSERT CODE HERE
	dim3 dimGrid(ceil((float)n / TILE_SZ_B), ceil((float)m / TILE_SZ_A), 1);
	dim3 dimBlock(1, TILE_SZ_A, 1);
	// Invoke CUDA kernel -----------------------------------------------------
	mysgemm<<<dimGrid, dimBlock>>>(m, n, k, A, B, C);
	//INSERT CODE HERE

}

